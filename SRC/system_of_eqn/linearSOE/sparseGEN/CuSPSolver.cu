/* ****************************************************************** **
**    OpenSees - Open System for Earthquake Engineering Simulation    **
**          Pacific Earthquake Engineering Research Center            **
**                                                                    **
**                                                                    **
** (C) Copyright 1999, The Regents of the University of California    **
** All Rights Reserved.                                               **
**                                                                    **
** Commercial use of this program without express permission of the   **
** University of California, Berkeley, is strictly prohibited.  See   **
** file 'COPYRIGHT'  in main directory for information on usage and   **
** redistribution,  and for a DISCLAIMER OF ALL WARRANTIES.           **
**                                                                    **
** Developed by:                                                      **
**   Frank McKenna (fmckenna@ce.berkeley.edu)                         **
**   Gregory L. Fenves (fenves@ce.berkeley.edu)                       **
**   Filip C. Filippou (filippou@ce.berkeley.edu)                     **
**                                                                    **
** ****************************************************************** */

// $Revision: 1.0 $
// $Date: 2014-07-14 20:23:22 $
// $Source: /usr/local/cvs/OpenSees/SRC/system_of_eqn/linearSOE/sparseGEN/CuSPSolver.cpp,v $

// Written: neallee@tju.edu.cn 
// Modified from XZ Lu's CuSPSolver for GPL
// Created: 14/05
//
// Description: This file contains the implementation for CuSPSolver

#include <CuSPSolver.h>
#include <cusp/blas.h>
#include <cusp/format.h>
#include <cusp/linear_operator.h>
#include <cusp/csr_matrix.h>
#include <cusp/monitor.h>
#include <cusp/exception.h>
#include <cusp/krylov/bicg.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/gmres.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/precond/ainv.h>
#include <cusp/precond/aggregation/smoothed_aggregation.h>
#include <cusp/precond/diagonal.h>

#include <Windows.h>

CuSPSolver::CuSPSolver(void) :SparseGenRowLinSolver(SOLVER_TAGS_CuSP)
{
  single = 0;
  error = 0;

//  HINSTANCE hDLL = LoadLibrary("CuSPSolver.dll");
//  if (hDLL) {
//    SolveFunc = (CUSPSOLVE)GetProcAddress(hDLL, "CuSPSolve");
//
//    if (!SolveFunc) {
//      error = 1;
//      return;
//    }
//  }
//  else {
//    error = 2;
//    return;
//  }

  this->maxInteration = 100000;
  this->relTolerance = 1e-6;
  this->preCond = 0;  // 0 - none; 1 - diagonal; 2 - ainv
  this->solver = 0; // 0 - bicg; 1 - bicgstab; 2 - cg; 3 - gmres
}

CuSPSolver::CuSPSolver(int maxInt, double relTol, int pre, int solv) :SparseGenRowLinSolver(SOLVER_TAGS_CuSP)
{
  single = 0;
  error = 0;

//  HINSTANCE hDLL = LoadLibrary("CuSPSolver.dll");
//  if (hDLL) {
//    SolveFunc = (CUSPSOLVE)GetProcAddress(hDLL, "CuSPSolve");
//
//    if (!SolveFunc) {
//      error = 1;
//      return;
//    }
//  }
//  else {
//    error = 2;
//    return;
//  }

  this->maxInteration = maxInt;
  this->relTolerance = relTol;
  this->preCond = pre;  // 0 - none; 1 - diagonal; 2 - ainv
  this->solver = solv;  // 0 - bicg; 1 - bicgstab; 2 - cg; 3 - gmres
}

CuSPSolver::~CuSPSolver(void)
{
}

int
CuSPSolver::setSize()
{
  n = theSOE->size;
  nnz = theSOE->nnz;

  return 0;
}

int
CuSPSolver::setLinearSOE(SparseGenRowLinSOE &theLinearSOE)
{
  theSOE = &theLinearSOE;
  return 0;
}

int
CuSPSolver::sendSelf(int cTAg, Channel &theChannel)
{
  // doing nothing
  return 0;
}

int
CuSPSolver::recvSelf(int cTag,
Channel &theChannel, FEM_ObjectBroker &theBroker)
{
  // nothing to do
  return 0;
}

int 
CuSPSolver::solve(void)
{
  if (theSOE == 0) {
    opserr << "WARNING SuperLU::solve(void)- ";
    opserr << " No LinearSOE object has been set\n";
    return -1;
  }

  n = theSOE->size;
  // check for quick return
  if (n == 0)
    return 0;

  nnz = theSOE->nnz;

  Bptr = theSOE->B;
  Aptr = theSOE->A;

  rowPtr = theSOE->rowStartA;
  colInd = theSOE->colA;

  // allocate storage for matrix A with NNZ non-zeros
  cusp::csr_matrix<int, double, cusp::host_memory> hostA(n, n, nnz);

  // initialize matrix entries on host
  for (int l = 0; l < nnz; l++) {
    hostA.values[l] = Aptr[l];
    hostA.column_indices[l] = colInd[l];
  }
  for (int k = 0; k < n+1; k++) {
    hostA.row_offsets[k] = rowPtr[k];
  }

  // copy to the device
  cusp::csr_matrix<int, double, cusp::device_memory> devA = hostA;

  // allocate storage for solution (x) and right hand side (b)
  cusp::array1d<double, cusp::device_memory> x(n, 0.0);
  cusp::array1d<double, cusp::device_memory> b(n);
  for (int k = 0; k < n; k++) {
//    x[k] = Xptr[k];
    b[k] = Bptr[k];
  }

  // set stopping criteria:
  //  iteration_limit    = 100
  //  relative_tolerance = 1e-6
  cusp::verbose_monitor<double> monitor(b, 1000, 1e-6);
  //cusp::default_monitor<double> monitor(b, 1000, 1e-6);
  //cusp::convergence_monitor<double> monitor(b, 1000, 1e-6);

  // set preconditioner (identity)
  cusp::identity_operator<double, cusp::device_memory> M(n, n);
  switch (preCond) // 0 - none; 1 - diagonal; 2 - ainv
  {
  case 0:
  {    
    break;
  }
  case 1:
  {
    cusp::precond::diagonal<double, cusp::device_memory> M(devA);
    break;
  }
  case 2:
  {
    //cusp::precond::scaled_bridson_ainv<double, cusp::device_memory> M(devA, 0.1);
    //cusp::precond:scaled_bridson_ainv<ValueType, MemorySpace> M(devA, 0, 10);
    cusp::precond::bridson_ainv<double, cusp::device_memory> M(devA, 0, -1, true, 2);
    break;
  }
  default:
  {
    opserr << "CuSPSolver::solve() - the wrong preCond type defined." << endln;
    break;
  }
  }
  
  // solve the linear system A x = b
  switch (solver) // 0 - bicg; 1 - bicgstab; 2 - cg; 3 - gmres
  {
  case 0:
  {
    // because both A and M are hermitian we can use 
    // them for their own conjugate transpose
    if (preCond != 0) cusp::krylov::bicgstab(devA, x, b, monitor, M);
    else cusp::krylov::bicgstab(devA, x, b);
    break;
  }
  case 1:
  {
    cusp::krylov::bicgstab(devA, x, b, monitor, M);
    break;
  }
  case 2:
  {
    cusp::krylov::cg(devA, x, b, monitor, M);
    break;
  }
  case 3:
  {
    int restart = 50;
    cusp::krylov::gmres(devA, x, b, restart, monitor, M);
    break;
  }
  default:
  {
    opserr << "CuSPSolver::solve() - the wrong solver type defined." << endln;
    break;
  }
  }

  for (int k = 0; k < n; k++) {
    theSOE->X[k] = x[k];
  }

  return 0;
}
